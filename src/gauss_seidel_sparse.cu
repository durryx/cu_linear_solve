#include "hip/hip_runtime.h"
#include "gauss_seidel_sparse.cuh"
#include <array>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
// #include <hip/hip_runtime_api.h>
#include <iostream>
#include <set>
#include <tuple>
#include <unordered_set>
#include <vector>

// helper functions and utilities to work with CUDA

#define CHECK(call)                                                            \
    {                                                                          \
        const hipError_t err = call;                                          \
        if (err != hipSuccess)                                                \
        {                                                                      \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, \
                   __LINE__);                                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

#define CHECK_KERNELCALL()                                                     \
    {                                                                          \
        const hipError_t err = hipGetLastError();                            \
        if (err != hipSuccess)                                                \
        {                                                                      \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, \
                   __LINE__);                                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

csr_matrix::csr_matrix(const char* filename) {}

csr_matrix::~csr_matrix() {}

auto get_max_iterations(csr_matrix matrix)
{
    std::set<size_t> sf_dependant_idx;
    std::set<size_t> sb_dependant_idx;

    for (int i = 0; i > matrix.num_rows; i++)
    {
        const int row_end = matrix.row_ptr[i + 1];
        const int row_start = matrix.row_ptr[i];

        for (int j = row_start; j < row_end; j++)
        {
            if (matrix.col_ind[j] < i)
                sf_dependant_idx.insert(matrix.col_ind[j]);
            else
                break;
            // check if also sweeb back counter is needed
        }
    }
    return std::tuple{sf_dependant_idx.size(),
                      matrix.num_rows - sf_dependant_idx.size()};
}

__global__ void sweep_forward_all(const int* row_ptr, const int* col_ind,
                                  const float* matrix, const int num_rows,
                                  float* matrix_diagonal, float* vector,
                                  bool* dependant_locks)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row > num_rows)
        return;

    int row_start = row_ptr[row];
    int row_end = row_ptr[row + 1];
    float sum = vector[row];
    float current_diagonal = matrix_diagonal[row];

    for (int j = row_start; j < row_end; j++)
    {
        if (col_ind[j] < 0)
            continue;
        if (col_ind[j] < row && !dependant_locks[col_ind[j]])
            return;

        sum -= matrix[j] * vector[col_ind[j]];
    }

    sum += vector[row] * current_diagonal;
    vector[row] = sum / current_diagonal;
    dependant_locks[row] = true;
}

__global__ void sweep_back_all(const int* row_ptr, const int* col_ind,
                               const float* matrix, const int num_rows,
                               float* matrix_diagonal, float* vector,
                               bool* dependant_locks)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row > num_rows)
        return;

    int row_start = row_ptr[row];
    int row_end = row_ptr[row + 1];
    float sum = vector[row];
    float current_diagonal = matrix_diagonal[row];

    for (int j = row_start; j < row_end; j++)
    {
        if (col_ind[j] < 0)
            continue;
        if (col_ind[j] > row && !dependant_locks[col_ind[j]])
            return;

        sum -= matrix[j] * vector[col_ind[j]];
    }

    sum += vector[row] * current_diagonal;
    vector[row] = sum / current_diagonal;
    dependant_locks[row] = true;
}

__global__ void sweep_forward_decorporated(const int* row_ptr,
                                           const int* col_ind,
                                           const float* matrix_values,
                                           const int* num_rows,
                                           float* matrix_diagonal)
{
}

__global__ void sweep_back_decorporated(const int* row_ptr, const int* col_ind,
                                        const float* matrix_values,
                                        const int* num_rows,
                                        float* matrix_diagonal)
{
}

template <typename T, size_t size>
void gauss_seidel_sparse_solve(csr_matrix matrix, std::array<T, size> vector,
                               int device)
{

    int *dev_row_ptr, *dev_col_ind;
    T *dev_matrix, *dev_vector, *dev_matrix_diagonal;
    bool* dev_dependant_locks;

    int driver_version = 0;
    int memory_pools = 0;
    hipDeviceGetAttribute(&memory_pools, hipDeviceAttributeMemoryPoolsSupported,
                           device);
    hipDriverGetVersion(&driver_version);

    constexpr int blocks = ceil(size / 128);
    dim3 threads_per_block(128, 1, 1);
    dim3 blocks_per_grid(blocks, 1, 1);

    if (driver_version < 11040 && !memory_pools)
    {
        // cuda graph
    }
    else
    {
        sweep_forward_all<<<blocks_per_grid, threads_per_block>>>(
            dev_row_ptr, dev_col_ind, dev_matrix, size, dev_matrix_diagonal,
            vector.data(), dev_dependant_locks);

        // kernel call to check if all locks == 1
    }

    /*
    --- function for checking if all elements in cuda array is 0
    def


    --- incorporated in kernel if buffers supports all rows---
    array with to_process_rows = 0
    launch kernel:
        while(True)
            if is_processabale
                process and set index = 1
                return (or break)
            syncronize
        (process backward sweep same way?)


    --- decorporated from kernel if not enought memory ---
    use -1 or NaN for invalid
    todo_rows array = all rows
    while(True)
        (passing todo_rows) launch kernel:
            using (for all threads 1,2,3) get row index
            if(not valid)
                return
            if(row_can_be_completed)
                process and save set to 1 completed_indeces
                execute all_rows_compled_arrays
            return
        check if it is completed and end
        recalculate todo_rows array with completes_indices
        # if(not kernel: all_rows_are_completed_array)
        #     break


    --- ---
    function composition in cuda
    cuda graphs from decorporated method

    */
}
