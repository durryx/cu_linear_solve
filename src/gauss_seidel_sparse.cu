#include "hip/hip_runtime.h"
#include "gauss_seidel_sparse.cuh"
#include <array>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
// #include <hip/hip_runtime_api.h>
#include <iostream>
#include <set>
#include <tuple>
#include <unordered_set>
#include <vector>

// helper functions and utilities to work with CUDA

#define CHECK(call)                                                            \
    {                                                                          \
        const hipError_t err = call;                                          \
        if (err != hipSuccess)                                                \
        {                                                                      \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, \
                   __LINE__);                                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

#define CHECK_KERNELCALL()                                                     \
    {                                                                          \
        const hipError_t err = hipGetLastError();                            \
        if (err != hipSuccess)                                                \
        {                                                                      \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, \
                   __LINE__);                                                  \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

csr_matrix::csr_matrix(const char* filename) {}

csr_matrix::~csr_matrix() {}

// iterate over columns to find unique indices less than row number
auto get_max_iterations(csr_matrix& matrix)
{
    std::set<size_t> sf_dependant_idx;
    std::set<size_t> sb_dependant_idx;

    for (int i = 0; i < matrix.num_rows; i++)
    {
        const int row_end = matrix.row_ptr[i + 1];
        const int row_start = matrix.row_ptr[i];

        for (int j = row_start; j < row_end; j++)
        {
            if (matrix.col_ind[j] < i)
                sf_dependant_idx.insert(matrix.col_ind[j]);
            else
                break;
            // check if also sweeb back counter is needed
        }
    }
    return sf_dependant_idx.size();
}

__global__ void count_indipendant_rows(const int* row_ptr, const int* col_ind,
                                       const float* matrix, const int num_rows,
                                       int* indipendant_rows)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row > num_rows)
        return;

    int row_start = row_ptr[row];
    // int row_end = row_ptr[row + 1];

    if (col_ind[row_start] < 0)
    {
        return;
        // check whether to row_start++
    }
    if (col_ind[row_start] >= row)
        atomicAdd(indipendant_rows, 1);
}

__global__ void sweep_forward_all(const int* row_ptr, const int* col_ind,
                                  const float* matrix, const int num_rows,
                                  float* matrix_diagonal, float* vector,
                                  bool* dependant_locks)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row > num_rows)
        return;

    int row_start = row_ptr[row];
    int row_end = row_ptr[row + 1];
    float sum = vector[row];
    float current_diagonal = matrix_diagonal[row];

    for (int j = row_start; j < row_end; j++)
    {
        if (col_ind[j] < 0)
            continue;
        if (col_ind[j] < row && !dependant_locks[col_ind[j]])
            return;

        sum -= matrix[j] * vector[col_ind[j]];
    }

    sum += vector[row] * current_diagonal;
    vector[row] = sum / current_diagonal;
    dependant_locks[row] = true;
}

__global__ void sweep_back_all(const int* row_ptr, const int* col_ind,
                               const float* matrix, const int num_rows,
                               float* matrix_diagonal, float* vector,
                               bool* dependant_locks)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row > num_rows)
        return;

    int row_start = row_ptr[row];
    int row_end = row_ptr[row + 1];
    float sum = vector[row];
    float current_diagonal = matrix_diagonal[row];

    for (int j = row_start; j < row_end; j++)
    {
        if (col_ind[j] < 0)
            continue;
        if (col_ind[j] > row && !dependant_locks[col_ind[j]])
            return;

        sum -= matrix[j] * vector[col_ind[j]];
    }

    sum += vector[row] * current_diagonal;
    vector[row] = sum / current_diagonal;
    dependant_locks[row] = true;
}

__global__ void sweep_forward_decorporated(const int* row_ptr,
                                           const int* col_ind,
                                           const float* matrix_values,
                                           const int* num_rows,
                                           float* matrix_diagonal)
{
}

__global__ void sweep_back_decorporated(const int* row_ptr, const int* col_ind,
                                        const float* matrix_values,
                                        const int* num_rows,
                                        float* matrix_diagonal)
{
}

template <typename T, size_t size>
void gauss_seidel_sparse_solve(csr_matrix matrix, std::array<T, size> vector,
                               int device)
{

    int *dev_row_ptr, *dev_col_ind, *dev_ind_rows;
    T *dev_matrix, *dev_vector, *dev_matrix_diagonal;
    bool* dev_dependant_locks;

    CHECK(hipMalloc(&dev_row_ptr, (matrix.num_rows + 1) * sizeof(int)));
    CHECK(hipMalloc(&dev_col_ind, matrix.num_vals * sizeof(int)));
    CHECK(hipMalloc(&dev_ind_rows, sizeof(int)));
    CHECK(hipMalloc(&dev_matrix, matrix.num_vals * sizeof(T)));
    CHECK(hipMalloc(&dev_vector, matrix.num_rows * sizeof(T)));
    CHECK(hipMalloc(&dev_matrix_diagonal, matrix.num_rows * sizeof(T)));
    CHECK(hipMalloc(&dev_dependant_locks, matrix.num_rows * sizeof(bool)));

    CHECK(hipMemcpy(dev_row_ptr, matrix.row_ptr,
                     (matrix.num_rows + 1) * sizeof(int),
                     hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_col_ind, matrix.col_ind, matrix.num_vals * sizeof(int),
                     hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_matrix, matrix.values, matrix.num_vals * sizeof(T),
                     hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_vector, vector.data(), matrix.num_rows * sizeof(T),
                     hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_matrix_diagonal, matrix.matrix_diagonal,
                     matrix.num_rows * sizeof(T), hipMemcpyHostToDevice));
    CHECK(hipMemset(dev_ind_rows, 0, sizeof(int)));
    CHECK(hipMemset(dev_dependant_locks, 0, matrix.num_rows * sizeof(bool)));

    int driver_version = 0;
    int memory_pools = 0;
    hipDeviceGetAttribute(&memory_pools, hipDeviceAttributeMemoryPoolsSupported,
                           device);
    hipDriverGetVersion(&driver_version);

    constexpr int blocks = ceil(size / 128);
    dim3 threads_per_block(128, 1, 1);
    dim3 blocks_per_grid(blocks, 1, 1);

    if (driver_version < 11040 && !memory_pools)
    {
        // cuda graph
    }
    else
    {
        count_indipendant_rows<<<blocks_per_grid, threads_per_block>>>(
            dev_row_ptr, dev_col_ind, dev_matrix, size, dev_ind_rows);
        CHECK_KERNELCALL();
        CHECK(hipDeviceSynchronize());

        int ind_rows = 0;
        CHECK(hipMemcpy(&ind_rows, dev_ind_rows, sizeof(int),
                         hipMemcpyDeviceToHost));
        // wrong !! should check for null rows, they are indipendant
        int tot_iterations = matrix.num_rows - 2 * ind_rows;

        for (; tot_iterations >= 0; tot_iterations--)
        {
            sweep_forward_all<<<blocks_per_grid, threads_per_block>>>(
                dev_row_ptr, dev_col_ind, dev_matrix, size, dev_matrix_diagonal,
                vector.data(), dev_dependant_locks);
        }

        // kernel call to check if all locks == 1
    }

    CHECK(hipMemcpy(vector.data(), dev_vector, matrix.num_rows * sizeof(T),
                     hipMemcpyDeviceToHost));

    /*
    --- function for checking if all elements in cuda array is 0
    def


    --- incorporated in kernel if buffers supports all rows---
    array with to_process_rows = 0
    launch kernel:
        while(True)
            if is_processabale
                process and set index = 1
                return (or break)
            syncronize
        (process backward sweep same way?)


    --- decorporated from kernel if not enought memory ---
    use -1 or NaN for invalid
    todo_rows array = all rows
    while(True)
        (passing todo_rows) launch kernel:
            using (for all threads 1,2,3) get row index
            if(not valid)
                return
            if(row_can_be_completed)
                process and save set to 1 completed_indeces
                execute all_rows_compled_arrays
            return
        check if it is completed and end
        recalculate todo_rows array with completes_indices
        # if(not kernel: all_rows_are_completed_array)
        #     break


    --- ---
    function composition in cuda
    cuda graphs from decorporated method

    */
}
